#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "vecadd.h"

__global__
void vecAddKernel(float* A, float *B, float* C, int n) {
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    if (i < n)
        C[i] = A[i] + B[i];
}

void vecAdd(float *A, float *B, float *C, int n) {
    int size = n * sizeof(float);
    float *d_A, *d_B, *d_C;

    hipMalloc((void **) &d_A, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_B, size);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_C, size);

    vecAddKernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}
