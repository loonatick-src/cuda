#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "matrixMul.h"

__global__ void matrixMulSquareKernel(float *d_M, float *d_N,
        float *d_P, int width) {
    // P_{ij} = M_ikN_kj
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if ((row < width && col < width)) {
        int k;
        int P_ind = row * width + col;
        d_P[P_ind] = 0.0;
        for (k = 0; k < width; k++) {
            d_P[P_ind] += d_M[row*width + k]*d_N[k*width + col];
        }
    }
}

void matrixMulSquare(float *M, float *N, float *P, int width) {
    const dim3 dimBlock(16, 16);
    const int numBlocks = ceil(width/16.0);
    const dim3 dimGrid(numBlocks, numBlocks);
    float *d_M, *d_N, *d_P;

    int sz = width*width * sizeof(float);

    hipMalloc((void **) &d_M, sz);
    hipMalloc((void **) &d_N, sz);
    hipMalloc((void **) &d_P, sz);

    hipMemcpy(d_M, M, sz, hipMemcpyHostToDevice);
    hipMemcpy(d_N, M, sz, hipMemcpyHostToDevice);
    
    matrixMulSquareKernel<<<dimGrid, dimBlock>>>(d_M,
            d_N, d_P, width);

    hipMemcpy(P, d_P, sz, hipMemcpyDeviceToHost);
    hipFree(d_M); hipFree(d_N); hipFree(d_P);
}
